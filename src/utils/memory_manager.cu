#include "memory_manager.h"
#include <iostream>
#include <mutex>
#include <algorithm>  // برای std::remove

MemoryManager& MemoryManager::getInstance() {
    static MemoryManager instance;
    return instance;
}

MemoryManager::~MemoryManager() {
    clearCache();
}

void* MemoryManager::allocate(size_t size, const std::string& tag) {
    void* ptr = nullptr;
    hipError_t err = hipMalloc(&ptr, size);
    if (err != hipSuccess) {
        std::cerr << "❌ Failed to allocate " << size << " bytes: " 
                  << hipGetErrorString(err) << std::endl;
        return nullptr;
    }
    
    std::lock_guard<std::mutex> lock(pool_mutex);
    memory_pool[tag].push_back(ptr);
    
    return ptr;
}

void MemoryManager::deallocate(void* ptr, const std::string& tag) {
    if (!ptr) return;
    
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        std::cerr << "❌ Failed to deallocate memory: " 
                  << hipGetErrorString(err) << std::endl;
    }
    
    std::lock_guard<std::mutex> lock(pool_mutex);
    auto& pool = memory_pool[tag];
    auto it = std::remove(pool.begin(), pool.end(), ptr);
    pool.erase(it, pool.end());
}

void* MemoryManager::allocateUnified(size_t size, const std::string& tag) {
    void* ptr = nullptr;
    hipError_t err = hipMallocManaged(&ptr, size);
    if (err != hipSuccess) {
        std::cerr << "❌ Failed to allocate unified memory: " 
                  << hipGetErrorString(err) << std::endl;
        return nullptr;
    }
    
    std::lock_guard<std::mutex> lock(pool_mutex);
    memory_pool[tag].push_back(ptr);
    
    return ptr;
}

hipError_t MemoryManager::copyToDevice(void* dst, const void* src, size_t size, hipStream_t stream) {
    return hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}

hipError_t MemoryManager::copyToHost(void* dst, const void* src, size_t size, hipStream_t stream) {
    return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, stream);
}

void* MemoryManager::getPreallocated(size_t size, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pool_mutex);
    auto& pool = memory_pool[tag];
    
    // جستجو برای حافظه با اندازه مناسب
    for (auto it = pool.begin(); it != pool.end(); ++it) {
        // در اینجا می‌توانیم اندازه حافظه را بررسی کنیم
        // برای سادگی، اولین حافظه موجود را برمی‌گردانیم
        void* ptr = *it;
        pool.erase(it);
        return ptr;
    }
    
    // اگر حافظه‌ای موجود نباشد، جدید تخصیص می‌دهیم
    return allocate(size, tag);
}

void MemoryManager::returnPreallocated(void* ptr, const std::string& tag) {
    std::lock_guard<std::mutex> lock(pool_mutex);
    memory_pool[tag].push_back(ptr);
}

void MemoryManager::printMemoryUsage() {
    size_t free, total;
    hipError_t err = hipMemGetInfo(&free, &total);
    if (err == hipSuccess) {
        std::cout << "📊 GPU Memory: " 
                  << (total - free) / (1024*1024) << " MB used, " 
                  << free / (1024*1024) << " MB free, " 
                  << total / (1024*1024) << " MB total" << std::endl;
    }
    
    std::cout << "📊 Memory pools:" << std::endl;
    for (const auto& pair : memory_pool) {
        std::cout << "  " << pair.first << ": " << pair.second.size() << " blocks" << std::endl;
    }
}

void MemoryManager::clearCache() {
    std::lock_guard<std::mutex> lock(pool_mutex);
    for (auto& pair : memory_pool) {
        for (void* ptr : pair.second) {
            hipFree(ptr);
        }
        pair.second.clear();
    }
    memory_pool.clear();
}