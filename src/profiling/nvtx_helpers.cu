#include "nvtx_helpers.h"
#include <iostream>

void nvtx_start_range(const char* name, uint32_t color) {
    nvtxEventAttributes_t eventAttrib = {0};
    eventAttrib.version = NVTX_VERSION;
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    eventAttrib.colorType = NVTX_COLOR_ARGB;
    eventAttrib.color = color;
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
    eventAttrib.message.ascii = name;
    nvtxRangePushEx(&eventAttrib);
}

void nvtx_end_range() {
    nvtxRangePop();
}

void nvtx_mark(const char* message, uint32_t color) {
    nvtxEventAttributes_t eventAttrib = {0};
    eventAttrib.version = NVTX_VERSION;
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
    eventAttrib.colorType = NVTX_COLOR_ARGB;
    eventAttrib.color = color;
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;
    eventAttrib.message.ascii = message;
    nvtxMarkEx(&eventAttrib);
}
